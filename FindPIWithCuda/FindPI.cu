#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include ""

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "Common.h"

#define MAX_THREADS_PER_BLOCK 1024

__global__ void preparePoints(const TPointData *input, TPointData *output, int count) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < count) {
		auto supportPoint = input[i];
		auto nextSupportPoint = input[(i + 1) % count];
		double nextPointX = (supportPoint.X + nextSupportPoint.X) / 2;
		double nextPointY = (supportPoint.Y + nextSupportPoint.Y) / 2;
		double radiusToNextPoint = sqrt(nextPointX * nextPointX + nextPointY * nextPointY);
		TPointData nextPoint;
		nextPoint.X = nextPointX / radiusToNextPoint;
		nextPoint.Y = nextPointY / radiusToNextPoint;
		output[i * 2] = supportPoint;
		output[(i * 2) + 1] = nextPoint;
	}
}

__global__ void calcLength(const TPointData *input, int count, double* accumulator) {
	__shared__ double local_lengths[MAX_THREADS_PER_BLOCK];
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < count)
	{
		auto prevPoint = input[i];
		auto point = input[(i + 1) % count];
		double diffX = point.X - prevPoint.X;
		double diffY = point.Y - prevPoint.Y;
		double length = sqrt(diffX * diffX + diffY * diffY);
		local_lengths[threadIdx.x] = length;
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		double local_sum = 0;
		for (int j = i, int local_index = 0; j < count && local_index < blockDim.x; j++, local_index++) {
			local_sum += local_lengths[local_index];
		}
		accumulator[blockIdx.x] = local_sum;
	}
}

/**
 * Program main
 */
int main(int argc, char **argv) {
    printf("[Finding PI Using CUDA] - Starting...\n");

	int i_count = 4;
	size_t i_size = i_count * sizeof(TPointData);
	TPointData *h_V = (TPointData *)malloc(i_size);

	h_V[0].X = 1;
	h_V[0].Y = 0;
	h_V[1].X = 0;
	h_V[1].Y = -1;
	h_V[2].X = -1;
	h_V[2].Y = 0;
	h_V[3].X = 0;
	h_V[3].Y = 1;

	// Allocate the device input vector
	TPointData *d_iV = NULL;
	checkCudaErrors(hipMalloc((void **)&d_iV, i_size));

	// Copy the host input vectors in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	checkCudaErrors(hipMemcpy(d_iV, h_V, i_size, hipMemcpyHostToDevice));


	for (int i = 0; i < 22; i++) {

		TPointData * d_oV = NULL;
		checkCudaErrors(hipMalloc((void **)&d_oV, i_size  * 2));

		int threadsPerBlock = i_count < MAX_THREADS_PER_BLOCK ? i_count : MAX_THREADS_PER_BLOCK;
		int blocksPerGrid = (i_count + threadsPerBlock - 1) / threadsPerBlock;
		printf("CUDA kernel launch with %d blocks of %d threads for %d items\n", blocksPerGrid, threadsPerBlock, i_count);

		preparePoints<<<blocksPerGrid, threadsPerBlock>>>(d_iV, d_oV, i_count);

		hipDeviceSynchronize();

		checkCudaErrors(hipGetLastError());

		checkCudaErrors(hipFree(d_iV));

		d_iV = d_oV;

		i_count = i_count * 2;
		i_size = i_size * 2;

		threadsPerBlock = i_count < MAX_THREADS_PER_BLOCK ? i_count : MAX_THREADS_PER_BLOCK;
		blocksPerGrid = (i_count + threadsPerBlock - 1) / threadsPerBlock;
		printf("CUDA kernel launch with %d blocks of %d threads for %d items\n", blocksPerGrid, threadsPerBlock, i_count);


		double result = 0;
		double *h_accumelator = NULL;

		h_accumelator = (double *)malloc(sizeof(double) * blocksPerGrid);

		double *d_accumulator = NULL;
		checkCudaErrors(hipMalloc((void **)&d_accumulator, sizeof(double) * blocksPerGrid));
		checkCudaErrors(hipMemset(d_accumulator, 0, sizeof(double) * blocksPerGrid));

		calcLength<<<blocksPerGrid, threadsPerBlock>>>(d_iV, i_count, d_accumulator);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		checkCudaErrors(hipMemcpy(h_accumelator, d_accumulator, sizeof(double) * blocksPerGrid, hipMemcpyDeviceToHost));

		for (int j = 0; j < blocksPerGrid; j++) {
			result += h_accumelator[j];
		}

		checkCudaErrors(hipFree(d_accumulator));
		free(h_accumelator);

		fprintf(stdout, "Expected Pi:	%1.16f\n", 3.1415926535897931);
		fprintf(stdout, "Calculated Pi:	%1.16f\n", result / 2);
	}

	// Free device global memory

	checkCudaErrors(hipFree(d_iV));

	free(h_V);
}

